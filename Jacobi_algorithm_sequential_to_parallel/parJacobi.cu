#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>


using namespace std;

static const int blockSize = 256;

void print_1D_array(int n, int N, double *a) {
	cout << "\nThe 1D array is" << endl;
	for (int i = 0; i < N; i++) {
		if (((i + 1) % n) == 1) {
			cout << endl;
		}
		cout << a[i] << "  ";
	}
	cout << endl;
}


/*************************************************************
* Unroll the last 6 iterations of the loop for each block
*************************************************************/
__device__
void warp(volatile double *partial_sum, int tid) {
	partial_sum[tid] += partial_sum[tid + 32];
	partial_sum[tid] += partial_sum[tid + 16];
	partial_sum[tid] += partial_sum[tid + 8];
	partial_sum[tid] += partial_sum[tid + 4];
	partial_sum[tid] += partial_sum[tid + 2];
	partial_sum[tid] += partial_sum[tid + 1];
}


/*************************************************************
* Performs parallel reduction (Binary Tree Structure)
*************************************************************/
__global__
void quickAdd(double *input, int N) {
	// Allocate shared memory
	__shared__ double partial_sum[blockSize];

	// Load elements AND do first add of reduction
	int index = blockIdx.x * (blockSize * 2) + threadIdx.x;
	int stride = gridDim.x * blockSize;

	for (int l = 0; l < blockSize; l++) {
		partial_sum[l] = 0;
	}
	__syncthreads();

	// Store first partial result instead of just the elements
	for (int k = index; k < N; k += stride) {
		if (k + blockSize < N) {
			partial_sum[threadIdx.x] += input[k] + input[k + blockSize];
			input[k] = 0;
			input[k + blockSize] = 0;
		}
		else {
			partial_sum[threadIdx.x] += input[k];
			input[k] = 0;
		}

	}
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockSize / 2; s > 32; s = s >> 1) {
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x < 32) {
		warp(partial_sum, threadIdx.x);
	}

	// Let the thread 0 of the block write it's result to main memory
	if (threadIdx.x == 0) {
		input[blockIdx.x] = partial_sum[0];
	}
}


/*************************************************************
* Performs one iteration of the Jacobi method and computes
* the residual value.
*************************************************************/
__global__
void one_jacobi_iteration(double xStart, double yStart, int maxXCount, int maxYCount,
	double *src, double *dst, double deltaX, double deltaY, double alpha, double omega, double *errors)
{
#define SRC(XX,YY) src[(YY)*maxXCount+(XX)]
#define DST(XX,YY) dst[(YY)*maxXCount+(XX)]
#define ERR(XX,YY) errors[(YY)*(maxXCount-2)+(XX)]

	double fX, fY;
	double updateVal;
	double f;
	// Coefficients
	double cx = 1.0 / (deltaX*deltaX);
	double cy = 1.0 / (deltaY*deltaY);
	double cc = -2.0*cx - 2.0*cy - alpha;

	int i_index = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i_stride = blockDim.x * gridDim.x;

	int j_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int j_stride = blockDim.y * gridDim.y;

	for (int y = j_index; y < maxYCount - 1; y = y + j_stride) {
		for (int x = i_index; x < maxXCount - 1; x = x + i_stride) {
			fY = yStart + (y - 1)*deltaY;
			fX = xStart + (x - 1)*deltaX;
			f = -alpha*(1.0 - fX*fX)*(1.0 - fY*fY) - 2.0*(1.0 - fX*fX) - 2.0*(1.0 - fY*fY);
			updateVal = ((SRC(x - 1, y) + SRC(x + 1, y))*cx + (SRC(x, y - 1) + SRC(x, y + 1))*cy + SRC(x, y)*cc - f) / cc;
			DST(x, y) = SRC(x, y) - omega*updateVal;
			ERR(x - 1, y - 1) = updateVal*updateVal;
		}
	}

}


/**********************************************************
* Checks the error between numerical and exact solutions
**********************************************************/
double checkSolution(double xStart, double yStart,
	int maxXCount, int maxYCount,
	double *u,
	double deltaX, double deltaY,
	double alpha)
{
#define U(XX,YY) u[(YY)*maxXCount+(XX)]
	int x, y;
	double fX, fY;
	double localError, error = 0.0;

	for (y = 1; y < (maxYCount - 1); y++)
	{
		fY = yStart + (y - 1)*deltaY;
		for (x = 1; x < (maxXCount - 1); x++)
		{
			fX = xStart + (x - 1)*deltaX;
			localError = U(x, y) - (1.0 - fX*fX)*(1.0 - fY*fY);
			error += localError*localError;
		}
	}
	return sqrt(error) / ((maxXCount - 2)*(maxYCount - 2));
}


int main(int argc, char **argv)
{
	int n, m, mits;
	double alpha, tol, relax;
	double maxAcceptableError;
	double error;
	double *u, *u_old, *tmp;
	double *d_u, *d_u_old;
	double *d_errors;
	int allocCount;
	int iterationCount, maxIterationCount;
	double t1, t2;
	double xLeft, yBottom, xRight, yUp;
	double deltaX, deltaY;
	int numBlocks;

	/*printf("Input n,m - grid dimension in x,y direction:\n");
	scanf("%d,%d", &n, &m);
	printf("Input alpha - Helmholtz constant:\n");
	scanf("%lf", &alpha);
	printf("Input relax - successive over-relaxation parameter:\n");
	scanf("%lf", &relax);
	printf("Input tol - error tolerance for the iterrative solver:\n");
	scanf("%lf", &tol);
	printf("Input mits - maximum solver iterations:\n");
	scanf("%d", &mits);*/

	n = 1680;
	m = n;
	alpha = 1.0;
	relax = 0.8;
	tol = 1e-15;
	mits = 50;

	printf("-> %d, %d, %g, %g, %g, %d\n", n, m, alpha, relax, tol, mits);

	allocCount = (n + 2)*(m + 2);
	// Those two calls also zero the boundary elements
	u = (double*)calloc(allocCount, sizeof(double)); //reverse order
	u_old = (double*)calloc(allocCount, sizeof(double));

	if (u == NULL || u_old == NULL)
	{
		printf("Not enough memory for two %ix%i matrices\n", n + 2, m + 2);
		exit(1);
	}
	maxIterationCount = mits;
	maxAcceptableError = tol;

	// Solve in [-1, 1] x [-1, 1]
	xLeft = -1.0, xRight = 1.0;
	yBottom = -1.0, yUp = 1.0;

	deltaX = (xRight - xLeft) / (n - 1);
	deltaY = (yUp - yBottom) / (m - 1);

	iterationCount = 0;
	error = HUGE_VAL;

	// Allocate Memory to GPU
	hipMalloc(&d_u, allocCount * sizeof(double));
	hipMalloc(&d_u_old, allocCount * sizeof(double));
	hipMalloc(&d_errors, n * m * sizeof(double));

	numBlocks = (n * m + blockSize - 1) / blockSize;

	//numBlocks = 1;

	dim3 block((int)sqrt(blockSize), (int)sqrt(blockSize), 1);
	dim3 grid((int)sqrt(numBlocks), (int)sqrt(numBlocks), 1);

	// copy to GPU
	hipMemcpy(d_u, u, allocCount * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_u_old, u_old, allocCount * sizeof(double), hipMemcpyHostToDevice);


	/********************************* ___Start timing___*********************************/
	clock_t start = clock(), diff;

	/* Iterate as long as it takes to meet the convergence criterion */
	while (iterationCount < maxIterationCount && error > maxAcceptableError)
	{
		// CUDA kernel function (one_jacobi_iteration) call
		one_jacobi_iteration << <grid, block >> >(xLeft, yBottom,
			n + 2, m + 2, d_u_old, d_u, deltaX, deltaY, alpha, relax, d_errors);

		// Swap the buffers
		tmp = d_u_old;
		d_u_old = d_u;
		d_u = tmp;

		// CUDA kernel function (quickAdd) call
		quickAdd << <numBlocks / 2, blockSize >> >(d_errors, n*m);
		quickAdd << <1, blockSize >> >(d_errors, n*m);

		hipMemcpy(&error, d_errors, sizeof(double), hipMemcpyDeviceToHost);

		error = sqrt(error) / (n * m);

		// Increment iteration 
		iterationCount++;
	}

	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;

	/*********************************___End timing___*********************************/


	// copy to Host
	hipMemcpy(u, d_u, allocCount * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(u_old, d_u_old, allocCount * sizeof(double), hipMemcpyDeviceToHost);

	printf("Iterations=%3d\n", iterationCount);
	printf("Time taken %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
	printf("Residual %e\n", error);
	//print_1D_array(n+2, (n+2)*(m+2), u_old);

	// u_old holds the solution after the most recent buffers swap
	double absoluteError = checkSolution(xLeft, yBottom,
		n + 2, m + 2,
		u_old,
		deltaX, deltaY,
		alpha);

	printf("The error of the iterative solution is %e\n", absoluteError);

	hipFree(d_u);
	hipFree(d_u_old);
	hipFree(d_errors);

	return 0;
}
