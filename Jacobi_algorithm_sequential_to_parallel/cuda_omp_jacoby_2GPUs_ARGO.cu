#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <omp.h>

using namespace std;

static const int blockSize = 256;

/*************************************************************
* Unroll the last 6 iterations of the loop for each block
*************************************************************/
__device__
void warp(volatile double *partial_sum, int tid) {
	partial_sum[tid] += partial_sum[tid + 32];
	partial_sum[tid] += partial_sum[tid + 16];
	partial_sum[tid] += partial_sum[tid + 8];
	partial_sum[tid] += partial_sum[tid + 4];
	partial_sum[tid] += partial_sum[tid + 2];
	partial_sum[tid] += partial_sum[tid + 1];
}


/*************************************************************
* Performs parallel reduction (Binary Tree Structure)
*************************************************************/
__global__
void quickAdd(double *input, int N) {
	// Allocate shared memory
	__shared__ double partial_sum[blockSize];

	// Load elements AND do first add of reduction
	int index = blockIdx.x * (blockSize * 2) + threadIdx.x;
	int stride = gridDim.x * blockSize;

	if (threadIdx.x == 0) {
		for (int l = 0; l < blockSize; l++) {
			partial_sum[l] = 0;
		}
	}
	__syncthreads();


	// Store first partial result instead of just the elements
	for (int k = index; k < N; k += stride) {
		if (k + blockSize < N) {
			partial_sum[threadIdx.x] += input[k] + input[k + blockSize];
			input[k] = 0;
			input[k + blockSize] = 0;
		}
		else {
			partial_sum[threadIdx.x] += input[k];
			input[k] = 0;
		}

	}
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockSize / 2; s > 32; s = s >> 1) {
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x < 32) {
		warp(partial_sum, threadIdx.x);
	}

	// Let the thread 0 of the block write it's result to main memory
	if (threadIdx.x == 0) {
		input[blockIdx.x] = partial_sum[0];
	}
}


/*************************************************************
* Performs one iteration of the Jacobi method and computes
* the residual value.
*************************************************************/
__global__
void one_jacobi_iteration(int tid, double xStart, double yStart, int maxXCount, int maxYCount,
	double *src, double *dst, double deltaX, double deltaY, double alpha, double omega, double *errors)
{
#define SRC(XX,YY) src[(YY)*maxXCount+(XX)-tid*(maxYCount-2)*maxXCount]
#define DST(XX,YY) dst[(YY)*maxXCount+(XX)-tid*(maxYCount-2)*maxXCount]
#define ERR(XX,YY) errors[(YY)*maxXCount+(XX)-tid*(maxYCount-2)*maxXCount]

	double fX, fY;
	double updateVal;
	double f;
	// Coefficients
	double cx = 1.0 / (deltaX*deltaX);
	double cy = 1.0 / (deltaY*deltaY);
	double cc = -2.0*cx - 2.0*cy - alpha;

	int i_index = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i_stride = blockDim.x * gridDim.x;

	int j_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int j_stride = blockDim.y * gridDim.y;

	for (int y = ((int)(tid * (maxYCount - 2))) + j_index; y < ((int)(tid * (maxYCount - 2))) + maxYCount - 1; y = y + j_stride) {
		for (int x = i_index; x < maxXCount - 1; x = x + i_stride) {
			fY = yStart + (y - 1)*deltaY;
			fX = xStart + (x - 1)*deltaX;
			f = -alpha*(1.0 - fX*fX)*(1.0 - fY*fY) - 2.0*(1.0 - fX*fX) - 2.0*(1.0 - fY*fY);
			updateVal = ((SRC(x - 1, y) + SRC(x + 1, y))*cx + (SRC(x, y - 1) + SRC(x, y + 1))*cy + SRC(x, y)*cc - f) / cc;
			DST(x, y) = SRC(x, y) - omega*updateVal;
			ERR(x, y) = updateVal*updateVal;
		}
	}

}


/**********************************************************
* Checks the error between numerical and exact solutions
**********************************************************/
double checkSolution(double xStart, double yStart,
	int maxXCount, int maxYCount,
	double *u,
	double deltaX, double deltaY,
	double alpha)
{
#define U(XX,YY) u[(YY)*maxXCount+(XX)]
	int x, y;
	double fX, fY;
	double localError, error = 0.0;

	for (y = 1; y < (maxYCount - 1); y++)
	{
		fY = yStart + (y - 1)*deltaY;
		for (x = 1; x < (maxXCount - 1); x++)
		{
			fX = xStart + (x - 1)*deltaX;
			localError = U(x, y) - (1.0 - fX*fX)*(1.0 - fY*fY);
			error += localError*localError;
		}
	}
	return sqrt(error) / ((maxXCount - 2)*(maxYCount - 2));
}


int main(int argc, char **argv)
{
	int n, m, mits;
	double alpha, tol, relax;
	double maxAcceptableError;
	double error, err[2];
	double *u, *u_old, *tmp;
	double *d_u_0, *d_u_old_0, *d_u_1, *d_u_old_1;
	double *d_errors_0, *d_errors_1;
	int allocCount, halfCount;
	int iterationCount, maxIterationCount;
	double t1, t2;
	double xLeft, yBottom, xRight, yUp;
	double deltaX, deltaY;
	int numBlocks_0, numBlocks_1;
	int TID;
	double zero = 0.0;

	/*printf("Input n,m - grid dimension in x,y direction:\n");
	scanf("%d,%d", &n, &m);
	printf("Input alpha - Helmholtz constant:\n");
	scanf("%lf", &alpha);
	printf("Input relax - successive over-relaxation parameter:\n");
	scanf("%lf", &relax);
	printf("Input tol - error tolerance for the iterrative solver:\n");
	scanf("%lf", &tol);
	printf("Input mits - maximum solver iterations:\n");
	scanf("%d", &mits);*/

	n = 1680;
	m = n;
	alpha = 1.0;
	relax = 0.8;
	tol = 1e-15;
	mits = 50;

	printf("-> %d, %d, %g, %g, %g, %d\n", n, m, alpha, relax, tol, mits);

	allocCount = (n + 2) * (m + 2);
	halfCount = (n + 2) * (int)((m + 2) / 2);

	// Those two calls also zero the boundary elements
	u = (double*)calloc(allocCount, sizeof(double));
	u_old = (double*)calloc(allocCount, sizeof(double));

	if (u == NULL || u_old == NULL)
	{
		printf("Not enough memory for two %ix%i matrices\n", n + 2, m + 2);
		exit(1);
	}
	maxIterationCount = mits;
	maxAcceptableError = tol;

	// Solve in [-1, 1] x [-1, 1]
	xLeft = -1.0, xRight = 1.0;
	yBottom = -1.0, yUp = 1.0;

	deltaX = (xRight - xLeft) / (n - 1);
	deltaY = (yUp - yBottom) / (m - 1);

	iterationCount = 0;
	error = HUGE_VAL;

	// CUDA device settings
	numBlocks_0 = (halfCount + blockSize - 1) / blockSize;
	numBlocks_1 = ((allocCount - halfCount) + blockSize - 1) / blockSize;

	dim3 block((int)sqrt(blockSize), (int)sqrt(blockSize), 1);
	dim3 grid_0((int)sqrt(numBlocks_0), (int)sqrt(numBlocks_0), 1);
	dim3 grid_1((int)sqrt(numBlocks_1), (int)sqrt(numBlocks_1), 1);


	/********************************* ___Start timing___*********************************/
	clock_t start = clock(), diff;


	/************************\|/-_0_-\|/****************************/
	hipSetDevice(0);

	// Allocate Memory to GPU 0
	hipMalloc(&d_u_0, (halfCount + n + 2) * sizeof(double));
	hipMalloc(&d_u_old_0, (halfCount + n + 2) * sizeof(double));
	hipMalloc(&d_errors_0, halfCount * sizeof(double));

	// copy to GPU 0
	hipMemcpy(d_u_0, u, (halfCount + n + 2) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_u_old_0, u_old, (halfCount + n + 2) * sizeof(double), hipMemcpyHostToDevice);


	/************************\|/-_1_-\|/****************************/
	hipSetDevice(1);

	// Allocate Memory to GPU 1
	hipMalloc(&d_u_1, (allocCount - halfCount + n + 2) * sizeof(double));
	hipMalloc(&d_u_old_1, (allocCount - halfCount + n + 2) * sizeof(double));
	hipMalloc(&d_errors_1, (allocCount - halfCount) * sizeof(double));

	// copy to GPU 1
	hipMemcpy(d_u_1, &u[halfCount - n - 2], (allocCount - halfCount + n + 2) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_u_old_1, &u_old[halfCount - n - 2], (allocCount - halfCount + n + 2) * sizeof(double), hipMemcpyHostToDevice);


	/* Iterate as long as it takes to meet the convergence criterion */
	while (iterationCount < maxIterationCount && error > maxAcceptableError)
	{
#pragma omp parallel num_threads(2)
		{
			TID = omp_get_thread_num();

			if (TID == 0) {
				hipSetDevice(0);

				// CUDA kernel function (one_jacobi_iteration) call
				one_jacobi_iteration << <grid_0, block >> >(0, xLeft, yBottom, n + 2, (m / 2) + 2,
					d_u_old_0, d_u_0, deltaX, deltaY, alpha, relax, d_errors_0);

				// CUDA kernel function (quickAdd) calls
				quickAdd << <numBlocks_0, blockSize >> >(d_errors_0, halfCount);
				quickAdd << <1, blockSize >> >(d_errors_0, halfCount);

				hipMemcpy(&err[0], d_errors_0, sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(d_errors_0, &zero, sizeof(double), hipMemcpyHostToDevice);

				// Swap the buffers of GPU 0
				tmp = d_u_old_0;
				d_u_old_0 = d_u_0;
				d_u_0 = tmp;

			}
			else {
				hipSetDevice(1);

				// CUDA kernel function (one_jacobi_iteration) call
				one_jacobi_iteration << <grid_1, block >> >(1, xLeft, yBottom, n + 2, ((m + 1) / 2) + 2,
					d_u_old_1, d_u_1, deltaX, deltaY, alpha, relax, d_errors_1);

				// CUDA kernel function (quickAdd) calls
				quickAdd << <numBlocks_1, blockSize >> >(d_errors_1, (allocCount - halfCount));
				quickAdd << <1, blockSize >> >(d_errors_1, (allocCount - halfCount));

				hipMemcpy(&err[1], d_errors_1, sizeof(double), hipMemcpyDeviceToHost);
				hipMemcpy(d_errors_1, &zero, sizeof(double), hipMemcpyHostToDevice);

				// Swap the buffers of GPU 0
				tmp = d_u_old_1;
				d_u_old_1 = d_u_1;
				d_u_1 = tmp;
			}
		}
#pragma omp barrier

		// copy to Host from GPU 1
		hipSetDevice(1);
		hipMemcpy(&u_old[halfCount - n - 2], d_u_old_1, (allocCount - halfCount + n + 2) * sizeof(double), hipMemcpyDeviceToHost);

		// copy to Host from GPU 0
		hipSetDevice(0);
		hipMemcpy(u_old, d_u_old_0, halfCount * sizeof(double), hipMemcpyDeviceToHost);

		// copy to GPU 0 from Host
		// hipSetDevice(0);
		hipMemcpy(d_u_old_0, u_old, (halfCount + n + 2) * sizeof(double), hipMemcpyHostToDevice);

		// copy to GPU 1 from Host
		hipSetDevice(1);
		hipMemcpy(d_u_old_1, &u_old[halfCount - n - 2], (allocCount - halfCount + n + 2) * sizeof(double), hipMemcpyHostToDevice);

		error = sqrt(err[0] + err[1]) / (n * m);

		// Increment iteration
		iterationCount++;
	}

	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;

	/*********************************___End timing___*********************************/


	printf("Iterations=%3d\n", iterationCount);
	printf("Time taken %d seconds %d milliseconds\n", msec / 1000, msec % 1000);
	printf("Residual %e\n", error);

	// u_old holds the solution after the most recent buffers swap
	double absoluteError = checkSolution(xLeft, yBottom,
		n + 2, m + 2,
		u_old,
		deltaX, deltaY,
		alpha);

	printf("The error of the iterative solution is %e\n", absoluteError);


	hipSetDevice(0);
	hipFree(d_u_0);
	hipFree(d_u_old_0);
	hipFree(d_errors_0);


	hipSetDevice(1);
	hipFree(d_u_1);
	hipFree(d_u_old_1);
	hipFree(d_errors_1);

	return 0;
}
