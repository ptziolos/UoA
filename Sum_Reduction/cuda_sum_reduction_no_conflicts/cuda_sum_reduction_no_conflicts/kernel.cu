#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>
#include <numeric>
//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

using namespace std;

using std::accumulate;
using std::generate;
using std::cout;
using std::vector;

#define SHMEM_SIZE 256

__global__ void sumReduction(int *v, int *v_r) {
	// Allocate shared memory
	__shared__ int partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements into shared memory
	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

int main() {
	// Vector size
	//int N = 1 << 16;
	int N = 400;

	size_t bytes = N * sizeof(int);

	// Host data
	vector<int> h_v(N);
	vector<int> h_v_r(N);

	// Initialize the input data
	//generate(begin(h_v), end(h_v), []() { return rand() % 10; });
	generate(begin(h_v), end(h_v), []() { return 1; });

	// Allocate device memory
	int *d_v, *d_v_r;
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);

	// Copy to device
	hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);

	// TB Size
	const int TB_SIZE = 256;

	// Grid Size (No padding)
	int GRID_SIZE = (N + TB_SIZE - 1) / TB_SIZE;

	// Call kernels
	sumReduction << <GRID_SIZE, TB_SIZE >> >(d_v, d_v_r);

	sumReduction << <1, TB_SIZE >> >(d_v_r, d_v_r);

	// Copy to host;
	hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);

	// Print the result
	assert(h_v_r[0] == std::accumulate(begin(h_v), end(h_v), 0));

	cout << h_v_r[0] << endl;
	cout << "COMPLETED SUCCESSFULLY\n";

	return 0;
}